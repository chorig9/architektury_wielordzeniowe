#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <cstdlib>
#include <iostream>
#include <omp.h>
#include "constants.hpp"
#include <array>
#include <hip/hip_runtime.h>

static constexpr int MAX_PER_REGION = (REGIONS_SINGLE * REGIONS_SINGLE / (MIN_MASS * MIN_MASS));

void check_error(hipError_t err, const char* msg)
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s :%s!\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

struct CudaBalls
{
    float *x;
    float *y;
    float *v_x;
    float *v_y;
    float *mass;
    int *index;

    int size;
};

__device__
bool is_collision(float x1, float y1, float r1, float x2, float y2, float r2)
{
    float sumRadius = r1 + r2;

    float xd = x1 - x2;
    float yd = y1 - y2;

    float sqrRadius = sumRadius * sumRadius;
    float distSqr = (xd * xd) + (yd * yd);

    return distSqr <= sqrRadius; 
}

struct wall_distance
{
    float l_wall, r_wall, t_wall, b_wall;
};

__device__
wall_distance wall_collision_point(float x, float y)
{
    return {x - LEFT_WALL,
            RIGHT_WALL - x,
            TOP_WALL - y,
            y - BOTTOM_WALL};
}

__device__
void collide_wall(float &x, float &y, float m, float &v_x, float &v_y)
{
    auto wall_dists = wall_collision_point(x, y);
    float collision_x = x;
    float collision_y = y;

    if (wall_dists.l_wall < m) {
        x = LEFT_WALL + m;
        collision_x = LEFT_WALL;
    } else if (wall_dists.r_wall < m) {
        x = RIGHT_WALL - m;
        collision_x = RIGHT_WALL;
    /* only consider collision with one wall */
    } else if (wall_dists.t_wall < m) {
        y = TOP_WALL - m;
        collision_y = TOP_WALL;
    } else if (wall_dists.b_wall < m) {
        y = BOTTOM_WALL + m;
        collision_y = BOTTOM_WALL;
    } else {
        return;
    }

    auto dot = v_x * (x - collision_x) + v_y * (y - collision_y);
    auto norm = (x - collision_x) * (x - collision_x) + (y - collision_y) * (y - collision_y);
    auto dot_norm = dot / norm;

    v_x -= 2 * dot_norm * (x - collision_x);
    v_y -= 2 * dot_norm * (y - collision_y);
}

__device__
void collide(float &x1, float &y1, float m1, float &v_x1, float &v_y1, float &x2, float& y2, float m2, float &v_x2, float &v_y2)
{
    if (!is_collision(x1, y1, m1, x2, y2, m2))
        return;

    float overlap_x = x2 - x1;
    float overlap_y = y2 - y1;

    float overlap_dist = sqrt(overlap_x * overlap_x + overlap_y * overlap_y);
    float radius_sum = m1 + m2;
    float coef = radius_sum / overlap_dist;

    /* move x2 away from x1 so that x2 - x1 == r1 + r2 */
    x2 = x1 + coef * overlap_x;
    y2 = y1 + coef * overlap_y;

    auto norm = (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2);
    norm = norm != 0 ? norm : 1;

    auto dot1 = (v_x1 - v_x2) * (x1 - x2) + (v_y1 - v_y2) * (y1 - y2);
    auto dot2 = (v_x2 - v_x1) * (x2 - x1) + (v_y2 - v_y1) * (y2 - y1);

    v_x1 -= (2 * m2) / (m1 + m2) * dot1 / norm * (x1 - x2);
    v_y1 -= (2 * m2) / (m1 + m2) * dot1 / norm * (y1 - y2);

    v_x2 -= (2 * m1) / (m1 + m2) * dot2 / norm * (x2 - x1);
    v_y2 -= (2 * m1) / (m1 + m2) * dot2 / norm * (y2 - y1);
}

__global__ void
clear_regions(int *regions_size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= REGIONS_NUM)
        return;

        regions_size[i] = 0;
}

__global__ void
split_to_regions(CudaBalls balls, int *regions, int *regions_size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= balls.size)
        return;

    int region_x = (balls.x[i] + WIDTH / 2) / REGIONS_SINGLE;
    int region_y = (balls.y[i] + HEIGHT / 2) / REGIONS_SINGLE;

    int r = region_x + region_y * REGIONS_X;
    int size = atomicAdd(&regions_size[r], 1);

    regions[r * MAX_PER_REGION + size] = i;
}

__global__ void
do_collide(CudaBalls balls, int *regions, int *regions_size)
{
    int r = blockIdx.x;
    int i = threadIdx.x;

    if (i >= regions_size[r])
        return;

    auto ball_index = regions[r * MAX_PER_REGION + i];

    int neighbour_regions[] = {1, REGIONS_X, REGIONS_X + 1};
    // neighbour regions
    for (int rs = 0; rs < 3; rs++) {
        int neighbour = r + neighbour_regions[rs];

        if (neighbour < 0 || neighbour >= REGIONS_NUM)
            continue;

        for (int j = 0; j < regions_size[neighbour]; j++) {
            auto nball_index = regions[neighbour * MAX_PER_REGION + j];
            collide(balls.x[ball_index],
                    balls.y[ball_index],
                    balls.mass[ball_index],
                    balls.v_x[ball_index],
                    balls.v_y[ball_index],
                    balls.x[nball_index],
                    balls.y[nball_index],
                    balls.mass[nball_index],
                    balls.v_x[nball_index],
                    balls.v_y[nball_index]);
        }
    }

    // same region
    for (int j = i + 1; j < regions_size[r]; j++) {
            auto nball_index = regions[r * MAX_PER_REGION + j];
            collide(balls.x[ball_index],
                    balls.y[ball_index],
                    balls.mass[ball_index],
                    balls.v_x[ball_index],
                    balls.v_y[ball_index],
                    balls.x[nball_index],
                    balls.y[nball_index],
                    balls.mass[nball_index],
                    balls.v_x[nball_index],
                    balls.v_y[nball_index]);
    }
}

__global__ void
collide_wall(CudaBalls _balls)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= _balls.size)
        return;

    collide_wall(_balls.x[i],
        _balls.y[i],
        _balls.mass[i],
        _balls.v_x[i],
        _balls.v_y[i]);
}

__global__ void
advance(CudaBalls balls)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= balls.size)
        return;

    balls.x[i] += balls.v_x[i];
    balls.y[i] += balls.v_y[i];
}

template <typename T>
static T* aligned_alloc(int n) {
    void* tmp = 0;

    auto alignment = sizeof(T) > sizeof(void*) ? sizeof(T) : sizeof(void*);
    if (posix_memalign(&tmp, alignment, sizeof(T) * n)) {
        throw std::bad_alloc();
    }
    return (T*)tmp;
}

struct Balls
{
    Balls(int num = MAX_PER_REGION): size(0), capacity(num) {
        x = aligned_alloc<float>(num);
        y = aligned_alloc<float>(num);
        v_x = aligned_alloc<float>(num);
        v_y = aligned_alloc<float>(num);
        mass = aligned_alloc<float>(num);
        index = aligned_alloc<int>(num);
    }

    ~Balls() {
        std::free(x);
        std::free(y);
        std::free(v_x);
        std::free(v_y);
        std::free(mass);
        std::free(index);
    }

    float *x;
    float *y;
    float *v_x;
    float *v_y;
    float *mass;
    int *index;

    int size;
    int capacity;

    char padding[8];
};

class simulation
{
public:
    simulation(int n_balls, int seed): _balls(n_balls)
    {
        srand(seed);

        x_regions = (WIDTH - 2 * MARGIN - 2 * MAX_MASS) / (2 * MAX_MASS);
        y_regions = (HEIGHT - 2 * MARGIN - 2 * MAX_MASS) / (2 * MAX_MASS);

        int n_regions = x_regions * y_regions;
        std::vector<bool> pos_bitmap(n_regions, false);

        for (int i = 0; i < n_balls; i++) {
            auto position = get_random_pos(pos_bitmap);

            _balls.x[i] = position.first;
            _balls.y[i] = position.second;
            _balls.v_x[i] = get_random(MIN_SPEED, MAX_SPEED);
            _balls.v_y[i] = get_random(MIN_SPEED, MAX_SPEED);
            _balls.mass[i] = get_random(MIN_MASS, MAX_MASS);
            _balls.index[i] = i;
        }

        _balls.size = n_balls;

        threadsPerBlock = 256;
        blocksPerGrid = (n_balls + threadsPerBlock - 1) / threadsPerBlock;

        regions = alloc_cuda_regions();
        regions_size = alloc_cuda_regions_size();
    }

    ~simulation()
    {
        hipError_t err = hipSuccess;

        err = hipFree(regions);
        check_error(err, "free regions");

        err = hipFree(regions_size);
        check_error(err, "free regions_size");
    }

    void step(int iters = 10)
    {
        CudaBalls c_balls = alloc_cuda_balls(_balls.size);
        memcpy_balls_to_device(c_balls, _balls);

        // #pragma omp for collapse(2)
        // for (int r1 = 0; r1 < REGIONS_X; r1 +=2) {
        //     for (int r2 = 0; r2 < REGIONS_X; r2 +=2) {
        //         do_collide(r1 * REGIONS_X + r2);
        //     }
        // }

        // #pragma omp for collapse(2)
        // for (int r1 = 1; r1 < REGIONS_X; r1 +=2) {
        //     for (int r2 = 0; r2 < REGIONS_X; r2 +=2) {
        //         do_collide(r1 * REGIONS_X + r2);
        //     }
        // }

        // #pragma omp for collapse(2)
        // for (int r1 = 0; r1 < REGIONS_X; r1 +=2) {
        //     for (int r2 = 1; r2 < REGIONS_X; r2 +=2) {
        //         do_collide(r1 * REGIONS_X + r2);
        //     }
        // }

        // #pragma omp for collapse(2)
        // for (int r1 = 1; r1 < REGIONS_X; r1 += 2) {
        //     for (int r2 = 1; r2 < REGIONS_X; r2 +=2) {
        //         do_collide(r1 * REGIONS_X + r2);
        //     }
        // }

        // do_collide<<REGIONS_X, 

        for (int i = 0; i < iters; i++) {
            clear_regions<<<blocksPerGrid, threadsPerBlock>>>(regions_size);
            split_to_regions<<<blocksPerGrid, threadsPerBlock>>>(c_balls, regions, regions_size);

            do_collide<<<REGIONS_NUM, 128>>>(c_balls, regions, regions_size);
            collide_wall<<<blocksPerGrid, threadsPerBlock>>>(c_balls);

            advance<<<blocksPerGrid, threadsPerBlock>>>(c_balls);
            hipError_t err = hipGetLastError();
            check_error(err, "count kernel");
        }

        memcpy_balls_to_host(c_balls, _balls);
        free_cuda_balls(c_balls);
    }

    const Balls& balls()
    {
        return _balls;
    }

private:
    int *alloc_cuda_regions()
    {
        int *regions;

        hipError_t err = hipSuccess;
        err = hipMalloc((void **)&regions, REGIONS_NUM * MAX_PER_REGION * sizeof(int));
        check_error(err, "table allocation regions");

        return regions;
    }

    int *alloc_cuda_regions_size()
    {
        int *regions_s;

        hipError_t err = hipSuccess;
        err = hipMalloc((void **)&regions_s, REGIONS_NUM * sizeof(int));
        check_error(err, "table allocation regions_s");

        return regions_s;
    }

    CudaBalls alloc_cuda_balls(int size)
    {
        CudaBalls c_balls;
        c_balls.size = size;

        hipError_t err = hipSuccess;
        err = hipMalloc((void **)&c_balls.x, size * sizeof(float));
        check_error(err, "table allocation x");
        err = hipMalloc((void **)&c_balls.y, size * sizeof(float));
        check_error(err, "table allocation y");
        err = hipMalloc((void **)&c_balls.v_x, size * sizeof(float));
        check_error(err, "table allocation vx");
        err = hipMalloc((void **)&c_balls.v_y, size * sizeof(float));
        check_error(err, "table allocation vy");
        err = hipMalloc((void **)&c_balls.mass, size * sizeof(float));
        check_error(err, "table allocation mass");
        err = hipMalloc((void **)&c_balls.index, size * sizeof(int));
        check_error(err, "table allocation index");

        return c_balls;
    }

    void memcpy_balls_to_device(CudaBalls &c_balls, Balls &_balls)
    {
        hipError_t err = hipSuccess;
        err = hipMemcpy(c_balls.x, _balls.x, _balls.size * sizeof(float), hipMemcpyHostToDevice);
        check_error(err, "table memcpy to device x");
        err = hipMemcpy(c_balls.y, _balls.y, _balls.size * sizeof(float), hipMemcpyHostToDevice);
        check_error(err, "table memcpy to device y");
        err = hipMemcpy(c_balls.v_x, _balls.v_x, _balls.size * sizeof(float), hipMemcpyHostToDevice);
        check_error(err, "table memcpy to device vx");
        err = hipMemcpy(c_balls.v_y, _balls.v_y, _balls.size * sizeof(float), hipMemcpyHostToDevice);
        check_error(err, "table memcpy to device vy");
        err = hipMemcpy(c_balls.mass, _balls.mass, _balls.size * sizeof(float), hipMemcpyHostToDevice);
        check_error(err, "table memcpy to device mass");
        err = hipMemcpy(c_balls.index, _balls.index, _balls.size * sizeof(int), hipMemcpyHostToDevice);
        check_error(err, "table memcpy to device index");
    }

    void memcpy_balls_to_host(CudaBalls &c_balls, Balls &_balls)
    {
        hipError_t err = hipSuccess;
        err = hipMemcpy(_balls.x, c_balls.x, _balls.size * sizeof(float), hipMemcpyDeviceToHost);
        check_error(err, "table memcpy to host x");
        err = hipMemcpy(_balls.y, c_balls.y, _balls.size * sizeof(float), hipMemcpyDeviceToHost);
        check_error(err, "table memcpy to host y");
        err = hipMemcpy(_balls.v_x, c_balls.v_x, _balls.size * sizeof(float), hipMemcpyDeviceToHost);
        check_error(err, "table memcpy to host vx");
        err = hipMemcpy(_balls.v_y, c_balls.v_y, _balls.size * sizeof(float), hipMemcpyDeviceToHost);
        check_error(err, "table memcpy to host vy");
        err = hipMemcpy(_balls.mass, c_balls.mass, _balls.size * sizeof(float), hipMemcpyDeviceToHost);
        check_error(err, "table memcpy to host mass");
        err = hipMemcpy(_balls.index, c_balls.index, _balls.size * sizeof(int), hipMemcpyDeviceToHost);
        check_error(err, "table memcpy to host index");
    }

    void free_cuda_balls(CudaBalls &c_balls)
    {
        hipError_t err = hipSuccess;

        err = hipFree(c_balls.x);
        check_error(err, "free x");
        err = hipFree(c_balls.y);
        check_error(err, "free y");
        err = hipFree(c_balls.v_x);
        check_error(err, "free vx");
        err = hipFree(c_balls.v_y);
        check_error(err, "free vy");
        err = hipFree(c_balls.mass);
        check_error(err, "free mass");
        err = hipFree(c_balls.index);
        check_error(err, "free index");
    }

    float get_random(float lo, float hi)
    {
        return lo + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(hi-lo)));
    }

    std::pair<float, float> get_random_pos(std::vector<bool> &pos_bitmap)
    {
            int pos_random;
            do {
                pos_random = rand() % pos_bitmap.size();
            } while (pos_bitmap[pos_random] == true);

            pos_bitmap[pos_random] = true;

            float x = (float) ((pos_random % x_regions) * 2 * MAX_MASS + 2 * MAX_MASS - WIDTH / 2); 
            float y = (float) ((pos_random / x_regions) * 2 * MAX_MASS + 2 * MAX_MASS - HEIGHT / 2);

            return {x, y};
    }

    int x_regions, y_regions;
    Balls _balls;

    int *regions;
    int *regions_size;

    int threadsPerBlock;
    int blocksPerGrid;
};

// int
// main(int argc, char *argv[])
// {
//     // Error code to check return values for CUDA calls
//     hipError_t err = hipSuccess;

//     size_t freem, totalm;
//     err = hipMemGetInfo(&freem, &totalm);
//     check_error(err, "get info");
//     printf("Opt1: Free memory %lu, total memory %lu\n", freem, totalm);

//     // Reset the device and exit
//     err = hipDeviceReset();
//     check_error(err, "device reset");

//     printf("Done\n");
//     return 0;
// }


